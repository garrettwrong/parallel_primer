#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

const int numstep=100;
const int neps=numstep-1;

/* This preprocessed file is located in 'input_data' from the git root.
   If using the provided Makefile from this directory, it should be found. */
#include "monthly_sunspots_data.h"

/* a larger data set */
/* #include "random_2d_data.h" */

__global__ void all_pairs_distances_kernel(double* X, int n, int d, double* D){
  int i, k;
  double xi, xj, tmp;

  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if( tidx >= n){
    return;
  }

  for(i=0; i<n; i++){

    /* compute norm in dim d,
       this is sort of stupid for d=1, alas.
    */
    tmp = 0;
    for(k=0; k<d; k++){
      xi = X[d*i + k];
      xj = X[d*tidx + k];
      /* square */
      tmp += (xi-xj) * (xi-xj);
    }
    D[i*n + tidx] = sqrtf(tmp);
  }
}



double* all_pairs_distances(double* X, int n, int d){
  double* D;
  hipMalloc(&D, n * n * sizeof(double));

  double* X_dev;
  hipMalloc(&X_dev, n * d * sizeof(double));
  hipMemcpy(X_dev, X, n * d * sizeof(double), hipMemcpyHostToDevice);

  int blocksz = 1024;
  int nblocks = (n + blocksz -1 ) / blocksz;

  all_pairs_distances_kernel<<<nblocks, blocksz>>>(X_dev, n, d, D);

  return D;
}

__global__ void min_max_kernel(double* D, int n, double* buf){
  int i;
  double minD, maxD, val;

  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if(tidx >= n){
    return;
  }

  /* compute min max of D*/
  /* We expect non diagonal elements to be non zero,
     though this is not "robust".
     I think using symmetry here would actually hurt performance...
     do you know why?
  */
  minD = D[1];
  maxD = D[1];

  for(i=0; i<n; i++){
    /* skip diagonals */
    if(tidx == i){
      continue;
    }
    val = D[i*n + tidx];

    if(val<minD){
      minD = val;
    }

    if(val>maxD){
      maxD = val;
    }
  }
  buf[tidx] = minD;
  buf[n + tidx] = maxD;

}

__global__ void generate_epsilons_kernel(double* D, int n, double* epsilons_dev, double minD, double maxD){
  double step;

  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if(tidx >= neps){
    return;
  }

  /* compute a step size */
  step = (maxD - minD) / neps;
  /* printf("step %f\n", step); */
  /* remember, we'll skip the first step, so (i+1), since it would be count of 0...*/
  epsilons_dev[tidx] = minD + step*(tidx+1);
}

double* generate_epsilons(double* D, int n){
  int i;
  double minD, maxD;

  /* compute min max of D*/
  double* buf = (double*)calloc(2*n, sizeof(double));
  double* buf_dev;
  hipMalloc(&buf_dev, 2*n*sizeof(double));

  int blocksz = 1024;
  int nblocks = (n + blocksz -1 ) / blocksz;

  min_max_kernel<<<blocksz, nblocks>>>(D, n, buf_dev);
  hipMemcpy(buf, buf_dev, 2*n*sizeof(double), hipMemcpyDeviceToHost);
  minD = buf[0];
  maxD = buf[n];
  for(i=1; i<n; i++){
    if(buf[i]<minD){
      minD = buf[i];
    }

    if(buf[n+i]>maxD){
      maxD = buf[n+i];
    }
  }
  /* printf("minxD %f maxD %f\n", minD, maxD); */

  /* allocate space for epsilon array */
  double* epsilons_dev;
  hipMalloc(&epsilons_dev, neps*sizeof(double));

  blocksz = 1024;
  nblocks = (neps + blocksz -1 ) / blocksz;

  generate_epsilons_kernel<<<blocksz, nblocks>>>(D, n, epsilons_dev, minD, maxD);

  hipFree(buf_dev);
  free(buf);

  return epsilons_dev;
}

__global__ void correlation_integrals_kernel(double* D, int n, double* epsilons, int* C){
  int i, cnt;
  double eps;

  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
  if( tidx >= n){
    return;
  }
  if( tidy >= neps){
    return;
  }

  eps = epsilons[tidy];
    /* loop through D, counting if closer than eps */
  cnt = 0;
  for(i=0; i<n; i++){
    if(D[i*n + tidx] < eps) {
      cnt++;
    }
    /* assign */
  }

  /* Each thread writes its partial count to a scratch buffer, */
  C[neps*tidx + tidy] = cnt;

}

__global__ void correlation_integrals_redux(int* C, int n){
  int i;

  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if( tidx >= neps){
    return;
  }
  /* then we'll sum that scratch buffer. */
  for(i=1; i<n; i++){
    C[tidx] += C[neps*i + tidx];
  }
}


int* correlation_integrals(double* D, int n, double* epsilons){

  /* we'll need to make some space and transfer the epsilons */
  double* epsilons_dev;
  hipMalloc(&epsilons_dev, neps*sizeof(double));
  hipMemcpy(epsilons_dev, epsilons, neps*sizeof(double), hipMemcpyHostToDevice);

  /* note C_dev is a much larger buffer for reduction sum scratch space*/
  int* C = (int*)calloc(neps, sizeof(int));
  int* C_dev;
  hipMalloc(&C_dev, neps*n*sizeof(int));

  dim3 blocksz(128,128);
  dim3 nblocks((n + blocksz.x - 1 ) / blocksz.x,
	       (neps + blocksz.y - 1 ) / blocksz.y);

  correlation_integrals_kernel<<<blocksz, nblocks>>>(D, n, epsilons_dev, C_dev);
  correlation_integrals_redux<<<blocksz.y, nblocks.y>>>(C_dev, n);

  /* copy results to host*/
  hipMemcpy(C, C_dev, neps*sizeof(int), hipMemcpyDeviceToHost);

  hipFree(C_dev);

  return C;
}


void write_file(double* epsilons, int* C){
  int n;

  FILE* fh = fopen("correlation_integrals.dat", "w");

  for(n=0; n<neps; n++){
    fprintf(fh, "%f %d\n", epsilons[n], C[n]);
  }

  fclose(fh);
}


double estimate_dimension(double* epsilons_dev, int* C){
  int i;
  double* X = (double*)calloc(neps, sizeof(double));
  double* Y = (double*)calloc(neps, sizeof(double));
  double xhat, yhat, num, den;
  double slope;
  /* double inter; */

  double* epsilons = (double*)calloc(neps, sizeof(double));
  hipMemcpy(epsilons, epsilons_dev, neps*sizeof(double), hipMemcpyDeviceToHost);

  write_file(epsilons, C);

  /* Since we don't have a real limit situation here,
     we'll truncate the tail of this dataset. */
  int n = (int)(0.5*neps);

  xhat = 0.;
  yhat = 0.;
  for(i=0; i<n; i++){
    X[i] = logf(epsilons[i]);
    Y[i] = logf(C[i]);

    xhat += X[i];
    yhat += Y[i];
  }

  xhat /= n;
  yhat /= n;
  /* printf("xhat yhat %f %f %d\n", xhat, yhat, n); */

  num = 0.;
  den = 0.;
  for(i=0; i<n; i++){
    num += X[i] * Y[i];
    den += X[i] * X[i];
  }

  num -= n * xhat * yhat;
  den -= n * xhat * xhat;

  slope = num / den;
  /*inter = yhat - slope * xhat; */

  free(X);
  free(Y);

  return slope;

}


int main(int argc, char** argv){
  double* D_dev;
  double* epsilons_dev;
  int* C;

  /* input_data and input_data_n are provided by the pre baked header at TOF. */
  D_dev = all_pairs_distances(input_data, input_data_n, input_data_dim);

  epsilons_dev = generate_epsilons(D_dev, input_data_n);

  C = correlation_integrals(D_dev, input_data_n, epsilons_dev);

  printf("Estimated Correlation Dimension: %f\n",
	 estimate_dimension(epsilons_dev, C));

  hipFree(D_dev);
  hipFree(epsilons_dev);
  free(C);

  return 0;
};
